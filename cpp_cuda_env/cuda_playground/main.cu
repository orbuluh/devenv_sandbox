// This program computes the sum of two vectors of length N
// By: Nick from CoffeeBeforeArch


#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <iostream>
#include <vector>

static void HandleError(hipError_t err, const char *file, int line) {
  if (err != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
    exit(EXIT_FAILURE);
  }
}
#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

// CUDA kernel for vector addition
// __global__ means this is called from the CPU, and runs on the GPU
__global__ void vectorAdd(const int *__restrict a, const int *__restrict b,
                          int *__restrict c, int N) {
  // Calculate global thread ID
  int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

  // Boundary check
  if (tid < N) c[tid] = a[tid] + b[tid];
}

// Check vector add result
void verify_result(std::vector<int> &a, std::vector<int> &b,
                   std::vector<int> &c) {
  for (size_t i = 0U; i < a.size(); i++) {
    assert(c[i] == a[i] + b[i]);
  }
}

int main() {
  // Array size of 2^16 (65536 elements)
  constexpr int N = 1 << 16;
  constexpr size_t bytes = sizeof(int) * N;

  // Vectors for holding the host-side (CPU-side) data
  std::vector<int> a;
  a.reserve(N);
  std::vector<int> b;
  b.reserve(N);
  std::vector<int> c;
  c.reserve(N);

  // Initialize random numbers in each array
  for (size_t i = 0U; i < N; i++) {
    a.push_back(rand() % 100);
    b.push_back(rand() % 100);
  }

  // Allocate memory on the device
  int *d_a, *d_b, *d_c;
  HANDLE_ERROR(hipMalloc(&d_a, bytes));
  HANDLE_ERROR(hipMalloc(&d_b, bytes));
  HANDLE_ERROR(hipMalloc(&d_c, bytes));

  // Copy data from the host to the device (CPU -> GPU)
  HANDLE_ERROR(hipMemcpy(d_a, a.data(), bytes, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(d_b, b.data(), bytes, hipMemcpyHostToDevice));

  // Threads per CTA (1024)
  int NUM_THREADS = 1 << 10;

  // CTAs per Grid
  // We need to launch at LEAST as many threads as we have elements
  // This equation pads an extra CTA to the grid if N cannot evenly be divided
  // by NUM_THREADS (e.g. N = 1025, NUM_THREADS = 1024)
  int NUM_BLOCKS = (N + NUM_THREADS - 1) / NUM_THREADS;

  // Launch the kernel on the GPU
  // Kernel calls are asynchronous (the CPU program continues execution after
  // call, but no necessarily before the kernel finishes)
  vectorAdd<<<NUM_BLOCKS, NUM_THREADS>>>(d_a, d_b, d_c, N);

  // Copy sum vector from device to host
  // cudaMemcpy is a synchronous operation, and waits for the prior kernel
  // launch to complete (both go to the default stream in this case).
  // Therefore, this cudaMemcpy acts as both a memcpy and synchronization
  // barrier.
  HANDLE_ERROR(hipMemcpy(c.data(), d_c, bytes, hipMemcpyDeviceToHost));

  // Check result for errors
  verify_result(a, b, c);

  // Free memory on device
  HANDLE_ERROR(hipFree(d_a));
  HANDLE_ERROR(hipFree(d_b));
  HANDLE_ERROR(hipFree(d_c));

  std::cout << "COMPLETED SUCCESSFULLY\n";

  return 0;
}